/*
 * File: kernel.cu
 * Project: cuda_src
 * Created Date: 06/06/2023
 * Author: Shun Suzuki
 * -----
 * Last Modified: 01/08/2023
 * Modified By: Shun Suzuki (suzuki@hapis.k.u-tokyo.ac.jp)
 * -----
 * Copyright (c) 2023 Shun Suzuki. All rights reserved.
 *
 */


#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include <cstdint>

#ifdef AUTD3_USE_SINGLE_FLOAT
#define makeAUTDComplex make_cuComplex
#else
#define makeAUTDComplex make_hipDoubleComplex
#endif

#ifdef AUTD3_USE_SINGLE_FLOAT
typedef float autd3_float_t;
typedef cuComplex autd3_complex_t;
#else
typedef double autd3_float_t;
typedef hipDoubleComplex autd3_complex_t;
#endif

__device__ autd3_float_t absc2(const autd3_complex_t x) { return x.x * x.x + x.y * x.y; }
__device__ autd3_float_t absc(const autd3_complex_t x) { return sqrt(absc2(x)); }
__device__ autd3_complex_t conj(const autd3_complex_t a) { return makeAUTDComplex(a.x, -a.y); }
__device__ autd3_complex_t mulc(const autd3_complex_t a, const autd3_complex_t b) {
  return makeAUTDComplex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}
__device__ autd3_complex_t mulcr(const autd3_complex_t a, const autd3_float_t b) { return makeAUTDComplex(a.x * b, a.y * b); }

__device__ autd3_complex_t divcr(const autd3_complex_t x, const autd3_float_t y) {
  const autd3_float_t r = x.x / y;
  const autd3_float_t i = x.y / y;
  return makeAUTDComplex(r, i);
}

__global__ void cu_gs_normalize_kernel(const autd3_complex_t *x, uint32_t n, autd3_complex_t *y) {
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= n) return;
  y[i] = mulcr(divcr(y[i], absc(y[i])), x[i].x);
}

__global__ void cu_gspat_normalize_kernel(const autd3_complex_t *x, const autd3_complex_t *y, uint32_t n, autd3_complex_t *z) {
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= n) return;
  z[i] = mulcr(divcr(y[i], absc(y[i])), x[i].x);
}

__global__ void cu_gspat_normalize2_kernel(const autd3_complex_t *x, const autd3_complex_t *y, uint32_t n, autd3_complex_t *z) {
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= n) return;
  z[i] = mulcr(divcr(y[i], absc2(y[i])), x[i].x * x[i].x);
}

__global__ void get_diagonal_kernel(const autd3_complex_t *a, uint32_t row, uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi) {
    unsigned int idx = yi + xi * row;
    b[xi] = a[idx];
  }
}

__global__ void get_diagonal_kernel(const autd3_float_t *a, uint32_t row, uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi) {
    unsigned int idx = yi + xi * row;
    b[xi] = a[idx];
  }
}

__global__ void set_diagonal_kernel_c(const autd3_complex_t *a, uint32_t n, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  if (xi >= n) return;
  unsigned int idx = xi + xi * n;
  b[idx] = a[xi];
}

__global__ void set_diagonal_kernel(const autd3_float_t *a, uint32_t n, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  if (xi >= n) return;
  unsigned int idx = xi + xi * n;
  b[idx] = a[xi];
}

__global__ void reciprocal_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  autd3_float_t s = absc2(a[idx]);
  const autd3_float_t x = a[idx].x / s;
  const autd3_float_t y = -a[idx].y / s;
  b[idx] = makeAUTDComplex(x, y);
}

__global__ void hadamard_product_kernel(const autd3_complex_t *a, const autd3_complex_t *b, const uint32_t row, const uint32_t col,
                                        autd3_complex_t *c) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  c[idx] = mulc(a[idx], b[idx]);
}

__global__ void abs_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = absc(a[idx]);
}

__global__ void sqrt_kernel(const autd3_float_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = sqrt(a[idx]);
}

__global__ void make_complex_kernel(const autd3_float_t *re, const autd3_float_t *im, const uint32_t row, const uint32_t col, autd3_complex_t *dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = makeAUTDComplex(re[idx], im[idx]);
}

__global__ void pow_kernel(const autd3_float_t *a, const autd3_float_t p, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = pow(a[idx], p);
}

__global__ void conj_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = conj(a[idx]);
}

__global__ void calc_singular_inv_kernel(autd3_float_t *d_s, uint32_t row, uint32_t col, autd3_float_t alpha, autd3_complex_t *p_singular_inv) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi)
    p_singular_inv[yi + xi * row] = makeAUTDComplex(d_s[xi] / (d_s[xi] * d_s[xi] + alpha), 0.0);
  else
    p_singular_inv[yi + xi * row] = makeAUTDComplex(0.0, 0.0);
}

__device__ autd3_complex_t expc(const autd3_complex_t x) {
  const autd3_float_t s = exp(x.x);
  const autd3_float_t r = cos(x.y);
  const autd3_float_t i = sin(x.y);
  return makeAUTDComplex(s * r, s * i);
}

__global__ void exp_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = expc(a[idx]);
}

__global__ void real_kernel(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = src[idx].x;
}
__global__ void imag_kernel(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = src[idx].y;
}

__global__ void col_sum_kernel(const autd3_float_t *din, uint32_t m, uint32_t n, autd3_float_t *dout) {
  uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row >= m) return;
  autd3_float_t sum = 0;
  for (uint32_t col = 0; col < n; col++) sum += din[col * m + row];
  dout[row] = sum;
}

#ifdef __cplusplus
extern "C" {
#endif

#define BLOCK_SIZE (32)

void cu_gs_normalize(const autd3_complex_t *x, const uint32_t len, autd3_complex_t *y) {
  unsigned int blocksPerGrid = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cu_gs_normalize_kernel<<<blocksPerGrid, BLOCK_SIZE>>>(x, len, y);
}

void cu_gspat_normalize(const autd3_complex_t *x, const autd3_complex_t *y, const uint32_t len, autd3_complex_t *z) {
  unsigned int blocksPerGrid = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cu_gspat_normalize_kernel<<<blocksPerGrid, BLOCK_SIZE>>>(x, y, len, z);
}

void cu_gspat_normalize2(const autd3_complex_t *x, const autd3_complex_t *y, const uint32_t len, autd3_complex_t *z) {
  unsigned int blocksPerGrid = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cu_gspat_normalize2_kernel<<<blocksPerGrid, BLOCK_SIZE>>>(x, y, len, z);
}

void cu_get_diagonal(const autd3_float_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  get_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_get_diagonal_c(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  get_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_set_diagonal(const autd3_float_t *a, const uint32_t n, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE * BLOCK_SIZE, 1, 1);
  dim3 grid((n - 1) / (BLOCK_SIZE * BLOCK_SIZE) + 1, 1, 1);
  set_diagonal_kernel<<<grid, block>>>(a, n, b);
}

void cu_set_diagonal_c(const autd3_complex_t *a, const uint32_t n, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE * BLOCK_SIZE, 1, 1);
  dim3 grid((n - 1) / (BLOCK_SIZE * BLOCK_SIZE) + 1, 1, 1);
  set_diagonal_kernel_c<<<grid, block>>>(a, n, b);
}

void cu_reciprocal(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  reciprocal_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_hadamard_product(const autd3_complex_t *a, const autd3_complex_t *b, const uint32_t row, const uint32_t col, autd3_complex_t *c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  hadamard_product_kernel<<<grid, block>>>(a, b, row, col, c);
}

void cu_abs(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  abs_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_sqrt(const autd3_float_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  sqrt_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_make_complex(const autd3_float_t *re, const autd3_float_t *im, const uint32_t row, const uint32_t col, autd3_complex_t *dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  make_complex_kernel<<<grid, block>>>(re, im, row, col, dst);
}

void cu_pow(const autd3_float_t *a, const autd3_float_t p, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  pow_kernel<<<grid, block>>>(a, p, row, col, b);
}

void cu_conj(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  conj_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_calc_singular_inv(autd3_float_t *d_s, const uint32_t row, const uint32_t col, const autd3_float_t alpha, autd3_complex_t *p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, row, col, alpha, p_singular_inv);
}

void cu_exp(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  exp_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_real(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  real_kernel<<<grid, block>>>(src, row, col, dst);
}

void cu_imag(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  imag_kernel<<<grid, block>>>(src, row, col, dst);
}

void cu_reduce_col(const autd3_float_t *mat, const uint32_t m, const uint32_t n, autd3_float_t *result) {
  dim3 block(1, BLOCK_SIZE * BLOCK_SIZE, 1);
  dim3 grid(1, (m - 1) / (BLOCK_SIZE * BLOCK_SIZE) + 1, 1);
  col_sum_kernel<<<grid, block>>>(mat, m, n, result);
}

#ifdef __cplusplus
}
#endif
