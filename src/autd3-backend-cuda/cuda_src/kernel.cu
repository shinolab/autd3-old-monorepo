/*
 * File: kernel.cu
 * Project: cuda_src
 * Created Date: 06/06/2023
 * Author: Shun Suzuki
 * -----
 * Last Modified: 10/08/2023
 * Modified By: Shun Suzuki (suzuki@hapis.k.u-tokyo.ac.jp)
 * -----
 * Copyright (c) 2023 Shun Suzuki. All rights reserved.
 *
 */


#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include <cstdint>

#ifdef AUTD3_USE_SINGLE_FLOAT
#define makeAUTDComplex make_cuComplex
#else
#define makeAUTDComplex make_hipDoubleComplex
#endif

#ifdef AUTD3_USE_SINGLE_FLOAT
typedef float autd3_float_t;
typedef cuComplex autd3_complex_t;
#else
typedef double autd3_float_t;
typedef hipDoubleComplex autd3_complex_t;
#endif

__device__ autd3_float_t absc2(const autd3_complex_t x) { return x.x * x.x + x.y * x.y; }
__device__ autd3_float_t absc(const autd3_complex_t x) { return sqrt(absc2(x)); }
__device__ autd3_complex_t conj(const autd3_complex_t a) { return makeAUTDComplex(a.x, -a.y); }
__device__ autd3_complex_t mulc(const autd3_complex_t a, const autd3_complex_t b) {
  return makeAUTDComplex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}
__device__ autd3_complex_t mulcr(const autd3_complex_t a, const autd3_float_t b) { return makeAUTDComplex(a.x * b, a.y * b); }

__device__ autd3_complex_t divcr(const autd3_complex_t x, const autd3_float_t y) {
  const autd3_float_t r = x.x / y;
  const autd3_float_t i = x.y / y;
  return makeAUTDComplex(r, i);
}

__global__ void normalize_kernel(const autd3_complex_t *x, uint32_t row, uint32_t col, autd3_complex_t *y) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;
  unsigned int i = yi + xi * row;
  y[i] = divcr(x[i], absc(x[i]));
}

__global__ void get_diagonal_kernel(const autd3_complex_t *a, uint32_t row, uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi) {
    unsigned int idx = yi + xi * row;
    b[xi] = a[idx];
  }
}

__global__ void get_diagonal_kernel(const autd3_float_t *a, uint32_t row, uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi) {
    unsigned int idx = yi + xi * row;
    b[xi] = a[idx];
  }
}

__global__ void set_diagonal_kernel_c(const autd3_complex_t *a, uint32_t n, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  if (xi >= n) return;
  unsigned int idx = xi + xi * n;
  b[idx] = a[xi];
}

__global__ void set_diagonal_kernel(const autd3_float_t *a, uint32_t n, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  if (xi >= n) return;
  unsigned int idx = xi + xi * n;
  b[idx] = a[xi];
}

__global__ void reciprocal_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  autd3_float_t s = absc2(a[idx]);
  const autd3_float_t x = a[idx].x / s;
  const autd3_float_t y = -a[idx].y / s;
  b[idx] = makeAUTDComplex(x, y);
}

__global__ void hadamard_product_kernel(const autd3_complex_t *a, const autd3_complex_t *b, const uint32_t row, const uint32_t col,
                                        autd3_complex_t *c) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  c[idx] = mulc(a[idx], b[idx]);
}

__global__ void abs_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = absc(a[idx]);
}

__global__ void sqrt_kernel(const autd3_float_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = sqrt(a[idx]);
}

__global__ void make_complex_kernel(const autd3_float_t *re, const uint32_t row, const uint32_t col, autd3_complex_t *dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = makeAUTDComplex(re[idx], 0);
}

__global__ void make_complex2_kernel(const autd3_float_t *re, const autd3_float_t *im, const uint32_t row, const uint32_t col, autd3_complex_t *dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = makeAUTDComplex(re[idx], im[idx]);
}

__global__ void pow_kernel(const autd3_float_t *a, const autd3_float_t p, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = pow(a[idx], p);
}

__global__ void conj_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = conj(a[idx]);
}

__global__ void calc_singular_inv_kernel(autd3_float_t *d_s, uint32_t row, uint32_t col, autd3_float_t alpha, autd3_complex_t *p_singular_inv) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi)
    p_singular_inv[yi + xi * row] = makeAUTDComplex(d_s[xi] / (d_s[xi] * d_s[xi] + alpha), 0.0);
  else
    p_singular_inv[yi + xi * row] = makeAUTDComplex(0.0, 0.0);
}

__device__ autd3_complex_t expc(const autd3_complex_t x) {
  const autd3_float_t s = exp(x.x);
  const autd3_float_t r = cos(x.y);
  const autd3_float_t i = sin(x.y);
  return makeAUTDComplex(s * r, s * i);
}

__global__ void exp_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = expc(a[idx]);
}

__global__ void real_kernel(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = src[idx].x;
}

__global__ void imag_kernel(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = src[idx].y;
}

__global__ void col_sum_kernel(const autd3_float_t *din, uint32_t m, uint32_t n, autd3_float_t *dout) {
  uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row >= m) return;
  autd3_float_t sum = 0;
  for (uint32_t col = 0; col < n; col++) sum += din[col * m + row];
  dout[row] = sum;
}

__global__ void generate_propagation_matrix_kernel(const autd3_float_t *positions, const autd3_float_t *foci, const autd3_float_t *wavenums,
                                                   const autd3_float_t attens, const uint32_t row, const uint32_t col, autd3_complex_t *dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  autd3_float_t xd = foci[3 * yi] - positions[3 * xi];
  autd3_float_t yd = foci[3 * yi + 1] - positions[3 * xi + 1];
  autd3_float_t zd = foci[3 * yi + 2] - positions[3 * xi + 2];
  autd3_float_t dist = sqrt(xd * xd + yd * yd + zd * zd);
  autd3_float_t r = exp(-dist * attens) / dist;
  autd3_float_t phase = -wavenums[xi] * dist;
  dst[yi + xi * row] = makeAUTDComplex(r * cos(phase), r * sin(phase));
}

#ifdef __cplusplus
extern "C" {
#endif

#define BLOCK_SIZE (32)

void cu_normalize(const autd3_complex_t *x, const uint32_t row, const uint32_t col, autd3_complex_t *y) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  normalize_kernel<<<grid, block>>>(x, row, col, y);
}

void cu_get_diagonal(const autd3_float_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  get_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_get_diagonal_c(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  get_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_set_diagonal(const autd3_float_t *a, const uint32_t n, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE * BLOCK_SIZE, 1, 1);
  dim3 grid((n - 1) / (BLOCK_SIZE * BLOCK_SIZE) + 1, 1, 1);
  set_diagonal_kernel<<<grid, block>>>(a, n, b);
}

void cu_set_diagonal_c(const autd3_complex_t *a, const uint32_t n, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE * BLOCK_SIZE, 1, 1);
  dim3 grid((n - 1) / (BLOCK_SIZE * BLOCK_SIZE) + 1, 1, 1);
  set_diagonal_kernel_c<<<grid, block>>>(a, n, b);
}

void cu_reciprocal(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  reciprocal_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_hadamard_product(const autd3_complex_t *a, const autd3_complex_t *b, const uint32_t row, const uint32_t col, autd3_complex_t *c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  hadamard_product_kernel<<<grid, block>>>(a, b, row, col, c);
}

void cu_abs(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  abs_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_sqrt(const autd3_float_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  sqrt_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_make_complex(const autd3_float_t *re, const uint32_t row, const uint32_t col, autd3_complex_t *dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  make_complex_kernel<<<grid, block>>>(re, row, col, dst);
}

void cu_make_complex2(const autd3_float_t *re, const autd3_float_t *im, const uint32_t row, const uint32_t col, autd3_complex_t *dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  make_complex2_kernel<<<grid, block>>>(re, im, row, col, dst);
}

void cu_pow(const autd3_float_t *a, const autd3_float_t p, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  pow_kernel<<<grid, block>>>(a, p, row, col, b);
}

void cu_conj(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  conj_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_calc_singular_inv(autd3_float_t *d_s, const uint32_t row, const uint32_t col, const autd3_float_t alpha, autd3_complex_t *p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, row, col, alpha, p_singular_inv);
}

void cu_exp(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  exp_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_real(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  real_kernel<<<grid, block>>>(src, row, col, dst);
}

void cu_imag(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  imag_kernel<<<grid, block>>>(src, row, col, dst);
}

void cu_reduce_col(const autd3_float_t *mat, const uint32_t m, const uint32_t n, autd3_float_t *result) {
  dim3 block(1, BLOCK_SIZE * BLOCK_SIZE, 1);
  dim3 grid(1, (m - 1) / (BLOCK_SIZE * BLOCK_SIZE) + 1, 1);
  col_sum_kernel<<<grid, block>>>(mat, m, n, result);
}

void cu_generate_propagation_matrix(const autd3_float_t *positions, const autd3_float_t *foci, const autd3_float_t *wavenums,
                                    const autd3_float_t attens, const uint32_t row, const uint32_t col, autd3_complex_t *dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  generate_propagation_matrix_kernel<<<grid, block>>>(positions, foci, wavenums, attens, row, col, dst);
}

#ifdef __cplusplus
}
#endif
