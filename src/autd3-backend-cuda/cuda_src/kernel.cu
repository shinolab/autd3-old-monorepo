#include "hip/hip_runtime.h"
/*
 * File: kernel.cu
 * Project: cuda_src
 * Created Date: 06/06/2023
 * Author: Shun Suzuki
 * -----
 * Last Modified: 08/06/2023
 * Modified By: Shun Suzuki (suzuki@hapis.k.u-tokyo.ac.jp)
 * -----
 * Copyright (c) 2023 Shun Suzuki. All rights reserved.
 *
 */

#include <hip/hip_complex.h>

#include <cstdint>

#ifdef AUTD3_USE_SINGLE_FLOAT
#define makeAUTDComplex make_hipComplex
#else
#define makeAUTDComplex make_hipDoubleComplex
#endif

#ifdef AUTD3_USE_SINGLE_FLOAT
typedef float autd3_float_t;
typedef hipComplex autd3_complex_t;
#else
typedef double autd3_float_t;
typedef hipDoubleComplex autd3_complex_t;
#endif

__device__ autd3_float_t absc2(const autd3_complex_t x) { return x.x * x.x + x.y * x.y; }
__device__ autd3_float_t absc(const autd3_complex_t x) { return sqrt(absc2(x)); }
__device__ autd3_complex_t conj(const autd3_complex_t a) { return makeAUTDComplex(a.x, -a.y); }
__device__ autd3_complex_t mulc(const autd3_complex_t a, const autd3_complex_t b) {
  return makeAUTDComplex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}
__device__ autd3_complex_t mulcr(const autd3_complex_t a, const autd3_float_t b) { return makeAUTDComplex(a.x * b, a.y * b); }

__device__ autd3_complex_t divcr(const autd3_complex_t x, const autd3_float_t y) {
  const autd3_float_t r = x.x / y;
  const autd3_float_t i = x.y / y;
  return makeAUTDComplex(r, i);
}

__global__ void cu_gs_normalize_kernel(const autd3_complex_t *x, uint32_t n, autd3_complex_t *y) {
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= n) return;
  y[i] = mulcr(divcr(y[i], absc(y[i])), x[i].x);
}

__global__ void cu_gspat_normalize_kernel(const autd3_complex_t *x, const autd3_complex_t *y, uint32_t n, autd3_complex_t *z) {
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= n) return;
  z[i] = mulcr(divcr(y[i], absc(y[i])), x[i].x);
}

__global__ void cu_gspat_normalize2_kernel(const autd3_complex_t *x, const autd3_complex_t *y, uint32_t n, autd3_complex_t *z) {
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= n) return;
  z[i] = mulcr(divcr(y[i], absc2(y[i])), x[i].x * x[i].x);
}

__global__ void get_diagonal_kernel(const autd3_complex_t *a, uint32_t row, uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi) {
    unsigned int idx = yi + xi * row;
    b[xi] = a[idx];
  }
}

__global__ void get_diagonal_kernel(const autd3_float_t *a, uint32_t row, uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi) {
    unsigned int idx = yi + xi * row;
    b[xi] = a[idx];
  }
}

__global__ void set_diagonal_kernel(const autd3_complex_t *a, uint32_t row, uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = xi == yi ? a[xi] : makeAUTDComplex(0.0, 0.0);
}

__global__ void set_diagonal_kernel(const autd3_float_t *a, uint32_t row, uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = xi == yi ? a[xi] : 0;
}

__global__ void reciprocal_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  autd3_float_t s = absc2(a[idx]);
  const autd3_float_t x = a[idx].x / s;
  const autd3_float_t y = -a[idx].y / s;
  b[idx] = makeAUTDComplex(x, y);
}

__global__ void hadamard_product_kernel(const autd3_complex_t *a, const autd3_complex_t *b, const uint32_t row, const uint32_t col,
                                        autd3_complex_t *c) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  c[idx] = mulc(a[idx], b[idx]);
}

__global__ void abs_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = absc(a[idx]);
}

__global__ void sqrt_kernel(const autd3_float_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = sqrt(a[idx]);
}

__global__ void make_complex_kernel(const autd3_float_t *re, const autd3_float_t *im, const uint32_t row, const uint32_t col, autd3_complex_t *dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = makeAUTDComplex(re[idx], im[idx]);
}

__global__ void pow_kernel(const autd3_float_t *a, const autd3_float_t p, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = pow(a[idx], p);
}

__global__ void conj_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = conj(a[idx]);
}

__global__ void calc_singular_inv_kernel(autd3_float_t *d_s, uint32_t row, uint32_t col, autd3_float_t alpha, autd3_complex_t *p_singular_inv) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi)
    p_singular_inv[yi + xi * row] = makeAUTDComplex(d_s[xi] / (d_s[xi] * d_s[xi] + alpha), 0.0);
  else
    p_singular_inv[yi + xi * row] = makeAUTDComplex(0.0, 0.0);
}

__device__ autd3_complex_t expc(const autd3_complex_t x) {
  const autd3_float_t s = exp(x.x);
  const autd3_float_t r = cos(x.y);
  const autd3_float_t i = sin(x.y);
  return makeAUTDComplex(s * r, s * i);
}

__global__ void exp_kernel(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = expc(a[idx]);
}

__global__ void real_kernel(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = src[idx].x;
}
__global__ void imag_kernel(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = src[idx].y;
}

__global__ void col_sum_kernel(const autd3_float_t *din, uint32_t m, uint32_t n, autd3_float_t *dout) {
  extern __shared__ autd3_float_t smem[];

  uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row >= m) return;

  uint32_t tid = threadIdx.x;
  uint32_t i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
  autd3_float_t local_sum = i < n ? din[i * m + row] : 0;
  if (i + blockDim.x < n) {
    local_sum += din[(i + blockDim.x) * m + row];
  }
  smem[tid] = local_sum;
  __syncthreads();

  for (unsigned int s = blockDim.x >> 1; s > 32; s >>= 1) {
    if (tid < s) {
      smem[tid] = local_sum = local_sum + smem[tid + s];
    }
    __syncthreads();
  }
  if (tid < 32) {
    if (blockDim.x >= 64) {
      local_sum += smem[tid + 32];
    }
    for (int offset = 32 >> 1; offset > 0; offset >>= 1) {
      local_sum += __shfl_down_sync(0xffffffff, local_sum, offset);
    }
  }
  if (tid == 0) {
    dout[blockIdx.x * m + row] = local_sum;
  }
}

#ifdef __cplusplus
extern "C" {
#endif

#define BLOCK_SIZE (32)

void cu_gs_normalize(const autd3_complex_t *x, const uint32_t len, autd3_complex_t *y) {
  unsigned int blocksPerGrid = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cu_gs_normalize_kernel<<<blocksPerGrid, BLOCK_SIZE>>>(x, len, y);
}

void cu_gspat_normalize(const autd3_complex_t *x, const autd3_complex_t *y, const uint32_t len, autd3_complex_t *z) {
  unsigned int blocksPerGrid = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cu_gspat_normalize_kernel<<<blocksPerGrid, BLOCK_SIZE>>>(x, y, len, z);
}

void cu_gspat_normalize2(const autd3_complex_t *x, const autd3_complex_t *y, const uint32_t len, autd3_complex_t *z) {
  unsigned int blocksPerGrid = (len + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cu_gspat_normalize2_kernel<<<blocksPerGrid, BLOCK_SIZE>>>(x, y, len, z);
}

void cu_get_diagonal(const autd3_float_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  get_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_get_diagonal_c(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  get_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_set_diagonal(const autd3_float_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  set_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_set_diagonal_c(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  set_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_reciprocal(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  reciprocal_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_hadamard_product(const autd3_complex_t *a, const autd3_complex_t *b, const uint32_t row, const uint32_t col, autd3_complex_t *c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  hadamard_product_kernel<<<grid, block>>>(a, b, row, col, c);
}

void cu_abs(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  abs_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_sqrt(const autd3_float_t *a, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  sqrt_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_make_complex(const autd3_float_t *re, const autd3_float_t *im, const uint32_t row, const uint32_t col, autd3_complex_t *dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  make_complex_kernel<<<grid, block>>>(re, im, row, col, dst);
}

void cu_pow(const autd3_float_t *a, const autd3_float_t p, const uint32_t row, const uint32_t col, autd3_float_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  pow_kernel<<<grid, block>>>(a, p, row, col, b);
}

void cu_conj(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  conj_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_calc_singular_inv(autd3_float_t *d_s, const uint32_t row, const uint32_t col, const autd3_float_t alpha, autd3_complex_t *p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, row, col, alpha, p_singular_inv);
}

void cu_exp(const autd3_complex_t *a, const uint32_t row, const uint32_t col, autd3_complex_t *b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  exp_kernel<<<grid, block>>>(a, row, col, b);
}

void cu_real(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  real_kernel<<<grid, block>>>(src, row, col, dst);
}

void cu_imag(const autd3_complex_t *src, const uint32_t row, const uint32_t col, autd3_float_t *dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  imag_kernel<<<grid, block>>>(src, row, col, dst);
}

uint32_t cu_reduce_col_buffer_size(const uint32_t m) { return m * BLOCK_SIZE / 2 * sizeof(autd3_float_t); }

void cu_reduce_col(const autd3_float_t *mat, const uint32_t m, const uint32_t n, autd3_float_t *result, autd3_float_t *buffer) {
  dim3 block(BLOCK_SIZE / 2, 1, 1);
  dim3 grid((n - 1) / BLOCK_SIZE + 1, m, 1);
  col_sum_kernel<<<grid, block, BLOCK_SIZE * sizeof(autd3_float_t)>>>(mat, m, n, buffer);
  col_sum_kernel<<<dim3(1, m, 1), dim3(max((grid.x + 1) / 2, 1), 1, 1), max(grid.x, 2) * sizeof(autd3_float_t)>>>(buffer, m, grid.x, result);
}

#ifdef __cplusplus
}
#endif
