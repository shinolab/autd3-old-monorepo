#include "hip/hip_runtime.h"
/*
 * File: kernel.cu
 * Project: cuda
 * Created Date: 13/05/2022
 * Author: Shun Suzuki
 * -----
 * Last Modified: 23/12/2022
 * Modified By: Shun Suzuki (suzuki@hapis.k.u-tokyo.ac.jp)
 * -----
 * Copyright (c) 2022 Shun Suzuki. All rights reserved.
 *
 */

#include <hip/hip_runtime_api.h>

#include <complex>

#include "./kernel.h"

#ifdef AUTD3_USE_SINGLE_FLOAT
#define makeAUTDComplex make_hipComplex
#else
#define makeAUTDComplex make_hipDoubleComplex
#endif

namespace autd3 {
namespace gain {
namespace holo {

__device__ autd3_float_t absc2(const autd3_complex_t x) { return x.x * x.x + x.y * x.y; }
__device__ autd3_float_t absc(const autd3_complex_t x) { return sqrt(absc2(x)); }
__device__ autd3_complex_t conj(const autd3_complex_t a) { return makeAUTDComplex(a.x, -a.y); }
__device__ autd3_complex_t mulc(const autd3_complex_t a, const autd3_complex_t b) {
  return makeAUTDComplex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

__global__ void abs_kernel(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = makeAUTDComplex(absc(a[idx]), 0.0);
}
__global__ void abs_kernel(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_float_t* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = absc(a[idx]);
}

void cu_abs(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  abs_kernel<<<grid, block>>>(a, row, col, b);
}
void cu_abs(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_float_t* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  abs_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void sqrt_kernel(const autd3_float_t* a, const uint32_t row, const uint32_t col, autd3_float_t* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = sqrt(a[idx]);
}

void cu_sqrt(const autd3_float_t* a, const uint32_t row, const uint32_t col, autd3_float_t* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  sqrt_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void conj_kernel(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = conj(a[idx]);
}

void cu_conj(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  conj_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void arg_kernel(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  const autd3_float_t s = absc(a[idx]);
  const autd3_float_t x = a[idx].x / s;
  const autd3_float_t y = a[idx].y / s;
  b[idx] = makeAUTDComplex(x, y);
}

void cu_arg(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  arg_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void reciprocal_kernel(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  autd3_float_t s = absc2(a[idx]);
  const autd3_float_t x = a[idx].x / s;
  const autd3_float_t y = -a[idx].y / s;
  b[idx] = makeAUTDComplex(x, y);
}

void cu_reciprocal(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  reciprocal_kernel<<<grid, block>>>(a, row, col, b);
}

__device__ autd3_complex_t expc(const autd3_complex_t x) {
  const autd3_float_t s = exp(x.x);
  const autd3_float_t r = cos(x.y);
  const autd3_float_t i = sin(x.y);
  return makeAUTDComplex(s * r, s * i);
}

__global__ void exp_kernel(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = expc(a[idx]);
}

void cu_exp(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  exp_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void pow_kernel(const autd3_float_t* a, const autd3_float_t p, const uint32_t row, const uint32_t col, autd3_float_t* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = pow(a[idx], p);
}

void cu_pow(const autd3_float_t* a, const autd3_float_t p, const uint32_t row, const uint32_t col, autd3_float_t* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  pow_kernel<<<grid, block>>>(a, p, row, col, b);
}

__global__ void imag_kernel(const autd3_complex_t* src, const uint32_t row, const uint32_t col, autd3_float_t* dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = src[idx].y;
}

void cu_imag(const autd3_complex_t* src, const uint32_t row, const uint32_t col, autd3_float_t* dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  imag_kernel<<<grid, block>>>(src, row, col, dst);
}

__global__ void real_kernel(const autd3_complex_t* src, const uint32_t row, const uint32_t col, autd3_float_t* dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = src[idx].x;
}

void cu_real(const autd3_complex_t* src, const uint32_t row, const uint32_t col, autd3_float_t* dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  real_kernel<<<grid, block>>>(src, row, col, dst);
}

__global__ void make_complex_kernel(const autd3_float_t* re, const autd3_float_t* im, const uint32_t row, const uint32_t col, autd3_complex_t* dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = makeAUTDComplex(re[idx], im[idx]);
}

void cu_make_complex(const autd3_float_t* re, const autd3_float_t* im, const uint32_t row, const uint32_t col, autd3_complex_t* dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  make_complex_kernel<<<grid, block>>>(re, im, row, col, dst);
}

__global__ void set_diagonal_kernel(const autd3_complex_t* a, uint32_t row, uint32_t col, autd3_complex_t* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = xi == yi ? a[xi] : makeAUTDComplex(0.0, 0.0);
}

void cu_set_diagonal(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  set_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void get_diagonal_kernel(const autd3_complex_t* a, uint32_t row, uint32_t col, autd3_complex_t* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi) {
    unsigned int idx = yi + xi * row;
    b[xi] = a[idx];
  }
}

void cu_get_diagonal(const autd3_complex_t* a, const uint32_t row, const uint32_t col, autd3_complex_t* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  get_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void get_diagonal_kernel(const autd3_float_t* a, uint32_t row, uint32_t col, autd3_float_t* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi) {
    unsigned int idx = yi + xi * row;
    b[xi] = a[idx];
  }
}

void cu_get_diagonal(const autd3_float_t* a, const uint32_t row, const uint32_t col, autd3_float_t* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  get_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void hadamard_product_kernel(const autd3_complex_t* a, const autd3_complex_t* b, const uint32_t row, const uint32_t col,
                                        autd3_complex_t* c) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  c[idx] = mulc(a[idx], b[idx]);
}

void cu_hadamard_product(const autd3_complex_t* a, const autd3_complex_t* b, const uint32_t row, const uint32_t col, autd3_complex_t* c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  hadamard_product_kernel<<<grid, block>>>(a, b, row, col, c);
}

__global__ void calc_singular_inv_kernel(autd3_float_t* d_s, uint32_t row, uint32_t col, autd3_float_t alpha, autd3_complex_t* p_singular_inv) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi)
    p_singular_inv[yi + xi * row] = makeAUTDComplex(d_s[xi] / (d_s[xi] * d_s[xi] + alpha), 0.0);
  else
    p_singular_inv[yi + xi * row] = makeAUTDComplex(0.0, 0.0);
}

void cu_calc_singular_inv(autd3_float_t* d_s, const uint32_t row, const uint32_t col, const autd3_float_t alpha, autd3_complex_t* p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, row, col, alpha, p_singular_inv);
}

__global__ void calc_singular_inv_kernel(autd3_float_t* d_s, uint32_t row, uint32_t col, autd3_float_t alpha, autd3_float_t* p_singular_inv) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi)
    p_singular_inv[yi + xi * row] = d_s[xi] / (d_s[xi] * d_s[xi] + alpha);
  else
    p_singular_inv[yi + xi * row] = 0.0;
}

void cu_calc_singular_inv(autd3_float_t* d_s, const uint32_t row, const uint32_t col, const autd3_float_t alpha, autd3_float_t* p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, row, col, alpha, p_singular_inv);
}

__global__ void col_sum_kernel(const autd3_float_t* din, uint32_t m, uint32_t n, autd3_float_t* dout) {
  extern __shared__ autd3_float_t smem[];

  uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row >= m) return;

  uint32_t tid = threadIdx.x;
  uint32_t i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
  autd3_float_t local_sum = i < n ? din[i * m + row] : 0;
  if (i + blockDim.x < n) {
    local_sum += din[(i + blockDim.x) * m + row];
  }
  smem[tid] = local_sum;
  __syncthreads();

  for (unsigned int s = blockDim.x >> 1; s > 32; s >>= 1) {
    if (tid < s) {
      smem[tid] = local_sum = local_sum + smem[tid + s];
    }
    __syncthreads();
  }
  if (tid < 32) {
    if (blockDim.x >= 64) {
      local_sum += smem[tid + 32];
    }
    for (int offset = 32 >> 1; offset > 0; offset >>= 1) {
      local_sum += __shfl_down_sync(0xffffffff, local_sum, offset);
    }
  }
  if (tid == 0) {
    dout[blockIdx.x * m + row] = local_sum;
  }
}

void cu_reduce_col(const autd3_float_t* mat, const uint32_t m, const uint32_t n, autd3_float_t* result, autd3_float_t* buffer) {
  dim3 block(BLOCK_SIZE / 2, 1, 1);
  dim3 grid((n - 1) / BLOCK_SIZE + 1, m, 1);
  col_sum_kernel<<<grid, block, BLOCK_SIZE * sizeof(autd3_float_t)>>>(mat, m, n, buffer);
  col_sum_kernel<<<dim3(1, m, 1), dim3(max((grid.x + 1) / 2, 1), 1, 1), max(grid.x, 2) * sizeof(autd3_float_t)>>>(buffer, m, grid.x, result);
}

}  // namespace holo
}  // namespace gain
}  // namespace autd3
